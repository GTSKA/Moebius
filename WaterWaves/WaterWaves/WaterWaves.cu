#include "hip/hip_runtime.h"
__global__ void createVertices(float4* positions, float time, unsigned int width, unsigned int height)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	// Calculate uv coordinates
	float u = x / (float)width; 
	float v = y / (float)height;
	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;

	// Calculate simple sine wave pattern
	float freq = 4.0f; 
	float w = sinf(u * freq + time) * cosf(v * freq + time) * 0.5f;

	//write positions
	positions[y*width + x] = make_float4(u, w, v, __int_as_float_(0xff00ff00));
}